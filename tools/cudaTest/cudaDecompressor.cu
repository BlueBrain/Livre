
#include "cudaDecompressorKernel.cu"

#include <cstdlib>
#include <cstdio>
#include <string.h>

#ifdef _WIN32
#  include <windows.h>
#endif

#include <cuda_gl_interop.h>

extern "C"
{

#define PRINT_TENSOR_TIMINGS

void checkCUDAError( const char *msg )
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err ));
        exit(EXIT_FAILURE);
    }
}


void allocateDeviceArray( void** array, int numBytes )
{
    hipMalloc( array, numBytes );
}


void deleteDeviceArray( void** array )
{
    hipFree( *array );
    *array = 0;
}


void copyArrayToDevice( void* device, const void* host, int numBytes )
{
    hipMemcpy( device, host, numBytes, hipMemcpyHostToDevice );
}

void copyArrayToHost( void* host, const void* device, int numBytes )
{
    hipMemcpy( host, device, numBytes, hipMemcpyDeviceToHost);
}


void registerGLBufferObject( unsigned int pbo )
{
    cudaGLRegisterBufferObject( pbo );
}


void unregisterGLBufferObject( unsigned int pbo )
{
    cudaGLUnregisterBufferObject( pbo );
}


void cuDecompTensor( unsigned char bytes,
                          void*    dst,
                    const float*   core,
                          float*   tmp1,
                          float*   tmp2,
                    const float*   u1,
                    const float*   u2,
                    const float*   u3,
                          int      coreDim,
                          int      blockDim,
                          int      uStride,
                          float    scale )
{
#ifdef PRINT_TENSOR_TIMINGS
    hipEvent_t start, stage1, stage2, stop;
    hipEventCreate( &start  );
    hipEventCreate( &stage1 );
    hipEventCreate( &stage2 );
    hipEventCreate( &stop   );
    hipEventRecord( start, 0 );
#endif

    dim3 threadD1( BLOCK_GRID, BLOCK_GRID );
    dim3 blockD1( ( coreDim+BLOCK_GRID-1)/BLOCK_GRID,
                  (blockDim+BLOCK_GRID-1)/BLOCK_GRID,
                    coreDim );
    cuDecompTensor_1<<< blockD1, threadD1 >>>( tmp1, core, u1, coreDim, blockDim, uStride );
#ifdef PRINT_TENSOR_TIMINGS
    hipEventRecord( stage1, 0 );
#endif

    dim3 threadD2( BLOCK_GRID, BLOCK_GRID );
    dim3 blockD2( (blockDim+BLOCK_GRID-1)/BLOCK_GRID,
                  (blockDim+BLOCK_GRID-1)/BLOCK_GRID,
                    coreDim );
    cuDecompTensor_2<<< blockD2, threadD2 >>>( tmp2, tmp1, u2, coreDim, blockDim, uStride );
#ifdef PRINT_TENSOR_TIMINGS
    hipEventRecord( stage2, 0 );
#endif

    dim3 threadD3( BLOCK_GRID, BLOCK_GRID );
    dim3 blockD3( (blockDim+BLOCK_GRID-1)/BLOCK_GRID,
                  (blockDim+BLOCK_GRID-1)/BLOCK_GRID,
                   blockDim );
    if( bytes == 1 )
        cuDecompTensor_3b<<< blockD3, threadD3 >>>( (unsigned  char*)dst, tmp2, u3, coreDim, blockDim, uStride, scale );
    else
        cuDecompTensor_3s<<< blockD3, threadD3 >>>( (unsigned short*)dst, tmp2, u3, coreDim, blockDim, uStride, scale );

#ifdef PRINT_TENSOR_TIMINGS
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    float elapsed1;
    float elapsed2;
    float elapsed3;
    float elapsedT;
    hipEventElapsedTime( &elapsed1, start,  stage1 );
    hipEventElapsedTime( &elapsed2, stage1, stage2 );
    hipEventElapsedTime( &elapsed3, stage2, stop   );
    hipEventElapsedTime( &elapsedT, start,  stop   );
    printf( " Time Total: %3.3f ms (%3.3f fps)\n", elapsedT, 1000.0f / elapsedT );
    printf( "     stage1: %3.3f ms (%3.3f fps)\n", elapsed1, 1000.0f / elapsed1 );
    printf( "     stage2: %3.3f ms (%3.3f fps)\n", elapsed2, 1000.0f / elapsed2 );
    printf( "     stage3: %3.3f ms (%3.3f fps)\n", elapsed3, 1000.0f / elapsed3 );
#endif
}



void _printDevProp( hipDeviceProp_t* devProp )
{
    printf("Major revision number:         %d\n",  devProp->major);
    printf("Minor revision number:         %d\n",  devProp->minor);
    printf("Name:                          %s\n",  devProp->name);
    printf("Total global memory:           %lu\n", devProp->totalGlobalMem);
    printf("Total shared memory per block: %lu\n", devProp->sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp->regsPerBlock);
    printf("Warp size:                     %d\n",  devProp->warpSize);
    printf("Maximum memory pitch:          %lu\n", devProp->memPitch);
    printf("Maximum threads per block:     %d\n",  devProp->maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of block: %d\n", i, devProp->maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of grid:  %d\n", i, devProp->maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp->clockRate);
    printf("Total constant memory:         %lu\n", devProp->totalConstMem);
    printf("Texture alignment:             %lu\n", devProp->textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp->deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp->multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp->kernelExecTimeoutEnabled ? "Yes" : "No"));
}

void printCUDAInfo()
{
    int devCount;
    hipGetDeviceCount( &devCount );
    printf( "CUDA Device Query...\n" );
    printf( "There are %d CUDA devices.\n", devCount );

    for(int i = 0; i < devCount; ++i )
    {
        printf ("\nCUDA Device #%d\n", i );
        hipDeviceProp_t devProp;
        hipGetDeviceProperties( &devProp, i );
        _printDevProp( &devProp );
    }
}




}









