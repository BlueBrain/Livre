
#include "tensorCUDAKernel.cu"

#include <cstdlib>
#include <cstdio>
#include <string.h>

#ifdef _WIN32
#  include <windows.h>
#endif

#include <cuda_gl_interop.h>

extern "C"
{

void checkCUDAError( const char *msg )
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf( stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err ));
        exit(EXIT_FAILURE);
    }
}


void allocateDeviceArray( void** array, int numBytes )
{
    hipMalloc( array, numBytes );
}


void deleteDeviceArray( void** array )
{
    hipFree( *array );
    *array = 0;
}


void copyArrayToDevice( void* device, const void* host, int numBytes )
{
    hipMemcpy( device, host, numBytes, hipMemcpyHostToDevice );
}

void copyArrayToHost( void* host, const void* device, int numBytes )
{
    hipMemcpy( host, device, numBytes, hipMemcpyDeviceToHost);
}


void registerGLBufferObject( unsigned int pbo )
{
    cudaGLRegisterBufferObject( pbo );
}


void unregisterGLBufferObject( unsigned int pbo )
{
    cudaGLUnregisterBufferObject( pbo );
}

void _printDevProp( hipDeviceProp_t* devProp )
{
    printf("Major revision number:         %d\n",  devProp->major);
    printf("Minor revision number:         %d\n",  devProp->minor);
    printf("Name:                          %s\n",  devProp->name);
    printf("Total global memory:           %lu\n", devProp->totalGlobalMem);
    printf("Total shared memory per block: %lu\n", devProp->sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp->regsPerBlock);
    printf("Warp size:                     %d\n",  devProp->warpSize);
    printf("Maximum memory pitch:          %lu\n", devProp->memPitch);
    printf("Maximum threads per block:     %d\n",  devProp->maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of block: %d\n",  i, devProp->maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of grid:  %d\n",  i, devProp->maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp->clockRate);
    printf("Total constant memory:         %lu\n", devProp->totalConstMem);
    printf("Texture alignment:             %lu\n", devProp->textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp->deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp->multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp->kernelExecTimeoutEnabled ? "Yes" : "No"));
}

void printCUDAInfo()
{
    int devCount;
    hipGetDeviceCount( &devCount );
    printf( "CUDA Device Query...\n" );
    printf( "There are %d CUDA devices.\n", devCount );

    for(int i = 0; i < devCount; ++i )
    {
        printf ("\nCUDA Device #%d\n", i );
        hipDeviceProp_t devProp;
        hipGetDeviceProperties( &devProp, i );
        _printDevProp( &devProp );
    }
}


void cudaDecompTensor( unsigned int  pboDst,
                          float*   tmp1,
                          float*   tmp2,
                    const float*   core,
                    const float*   u1,
                    const float*   u2,
                    const float*   u3,
                          int      coreDim,
                          int      blockDim,
                          int      uStride,
                          float    scale,
                          unsigned char bytes )
{
//    printCUDAInfo();

//    fprintf( stdout, "coreDim: %i, blockDim: %i, uStride: %i, BLOCK_GRID: %i \n", coreDim, blockDim, uStride, BLOCK_GRID );

    dim3 threadD1( BLOCK_GRID, BLOCK_GRID );
    dim3 blockD1( ( coreDim+BLOCK_GRID-1)/BLOCK_GRID,
                  (blockDim+BLOCK_GRID-1)/BLOCK_GRID,
                    coreDim );
    cuDecompTensor_1<<< blockD1, threadD1 >>>( tmp1, core, u1, coreDim, blockDim, uStride );


    dim3 threadD2( BLOCK_GRID, BLOCK_GRID );
    dim3 blockD2( (blockDim+BLOCK_GRID-1)/BLOCK_GRID,
                  (blockDim+BLOCK_GRID-1)/BLOCK_GRID,
                    coreDim );
    cuDecompTensor_2<<< blockD2, threadD2 >>>( tmp2, tmp1, u2, coreDim, blockDim, uStride );

    dim3 threadD3( BLOCK_GRID, BLOCK_GRID );
    dim3 blockD3( (blockDim+BLOCK_GRID-1)/BLOCK_GRID,
                  (blockDim+BLOCK_GRID-1)/BLOCK_GRID,
                   blockDim );

    void* dst;
    checkCUDAError( "before map object" );
    cudaGLMapBufferObject( &dst, pboDst );
    checkCUDAError( "after map object" );

    if( bytes == 1 )
        cuDecompTensor_3b<<< blockD3, threadD3 >>>( (unsigned  char*)dst, tmp2, u3, coreDim, blockDim, uStride, scale );
    else
    if( bytes == 2 )
        cuDecompTensor_3s<<< blockD3, threadD3 >>>( (unsigned short*)dst, tmp2, u3, coreDim, blockDim, uStride, scale );
    else
        fprintf( stderr, "ERROR: Only 1 and 2 bytes size is supported!" );

    cudaGLUnmapBufferObject( pboDst );
    checkCUDAError( "after unmap object" );
}

void cudaDecompTensorQuantized(
                            unsigned int  pboDst,
                            float*   tmp1,
                            float*   tmp2,
                        const uint8_t* core,
                        const float*   u1,
                        const float*   u2,
                        const float*   u3,
                            int      coreDim,
                            int      rank,
                            int      blockDim,
                            int      uStride,
                            float    scale,
                        unsigned char bytes )
{
    dim3 threads( BLOCK_GRID, BLOCK_GRID );
    dim3 blockD1( ( rank+BLOCK_GRID-1)/BLOCK_GRID,
                  (blockDim+BLOCK_GRID-1)/BLOCK_GRID,
                    rank );
    cuDecompTensor_quantized_1<<< blockD1, threads >>>( tmp1, core, u1, coreDim, rank, scale, blockDim, uStride );

    dim3 blockD2( (blockDim+BLOCK_GRID-1)/BLOCK_GRID,
                  (blockDim+BLOCK_GRID-1)/BLOCK_GRID,
                    rank );
    cuDecompTensor_2<<< blockD2, threads >>>( tmp2, tmp1, u2, rank, blockDim, uStride );

    dim3 blockD3( (blockDim+BLOCK_GRID-1)/BLOCK_GRID,
                  (blockDim+BLOCK_GRID-1)/BLOCK_GRID,
                   blockDim );
    void* dst;
    checkCUDAError( "before map object" );
    cudaGLMapBufferObject( &dst, pboDst );
    checkCUDAError( "after map object" );

    if( bytes == 1 )
        cuDecompTensor_3b<<< blockD3, threads >>>( (unsigned  char*)dst, tmp2, u3, rank, blockDim, uStride, 1.0 );
    else
    if( bytes == 2 )
        cuDecompTensor_3s<<< blockD3, threads >>>( (unsigned short*)dst, tmp2, u3, rank, blockDim, uStride, 1.0 );
    else
        fprintf( stderr, "ERROR: Only 1 and 2 bytes size is supported!" );

    cudaGLUnmapBufferObject( pboDst );
    checkCUDAError( "after unmap object" );
}


}
